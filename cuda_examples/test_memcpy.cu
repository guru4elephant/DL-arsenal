
int main()
{
    const int blockSize = 256, nStreams = 4;
    const unsigned int N = 1048576;
    const unsigned int bytes = N * sizeof(int);
    int *h_a = (int*)malloc(bytes);
    int *d_a;
    hipMalloc((int**)&d_a, bytes);

    memset(h_a, 0, bytes);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);

    return 0;
}

int main()
{
    const int blockSize = 256, nStreams = 4;
    const int n = 4 * 1024 * blockSize;
    const int streamSize = n / nStreams;
    const int streamBytes = streamSize * sizeof(float);
    const int bytes = n * sizeof(float);
    float *a, *d_a;
    
    hipStream_t stream[nStream];
    for (int i = 0; i < nStreams; ++i) {
        checkCuda(hipStreamCreate(&stream[i]));
    }
    memset(a, 0, bytes);
    

    checkCuda(hipHostMalloc((void**)&a, bytes));
    checkCuda(hipMalloc((void**)&d_a, bytes));
    for (int i = 0; i < nStreams; ++i) {
        checkCuda(hipMemcpyAsync());
        checkCuda()
    }
}